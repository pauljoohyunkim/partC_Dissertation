#include "hip/hip_runtime.h"
#include <cstdio>
#include "solver.cuh"
#include "vector.cuh"
#include "tpe.cuh"

__device__ void dkalphabeta(double* dev_blocks, int p, int q, int r, int k, unsigned int N, Vector& res, double alpha, double beta)
{
    p = p % (int) N;
    q = q % (int) N;
    r = r % (int) N;
    k = k % (int) N;
    int km1 = ((k-1) % (int) N + N) % (int) N;
    double xi;
    double eta;
    Vector dxi;
    Vector deta;

    if (p == k && r == k)
    {
        kjk(dev_blocks, p, q, r, N, xi, eta, dxi, deta, alpha, beta);
        printf("kjk\n");
    }
    else if (r == k)
    {
        ijk(dev_blocks, p, q, r, N, xi, eta, dxi, deta, alpha, beta);
        printf("ijk\n");
    }
    else if (p == km1 && r == km1)
    {
        km1jkm1(dev_blocks, p, q, r, N, xi, eta, dxi, deta, alpha, beta);
        printf("km1jkm1\n");
    }
    else if (p == k && r == km1)
    {
        kjkm1(dev_blocks, p, q, r, N, xi, eta, dxi, deta, alpha, beta);
        printf("kjkm1\n");
    }
    else if (q == k)
    {
        ikj(dev_blocks, p, q, r, N, xi, eta, dxi, deta, alpha, beta);
        printf("ikj\n");
    }
    else
    {
        printf("(p,q,r) tuple not defined\n");
    }

    res = (alpha / 2 * pow(xi, alpha/2-1) * dxi * eta - pow(xi, alpha/2) * deta) / pow(eta,2);
}

__device__ void kjk(double* dev_blocks, int p, int q, int r, unsigned int N, double& xi, double&eta, Vector& dxi, Vector& deta, double alpha, double beta)
{
    int k { p };
    int j { q };
    Vector xkEdge = vectorFromTensor(dev_blocks, k+1, N) - vectorFromTensor(dev_blocks, k, N);
    double xkEdgeLen = xkEdge.norm();
    Vector xkj = vectorFromTensor(dev_blocks, k, N) - vectorFromTensor(dev_blocks, j, N);
    double xkjLen = xkj.norm();

    xi = pow(xkEdgeLen * xkjLen, 2) - pow(xkEdge * xkj, 2);
    eta = pow(xkjLen, beta) * pow(xkEdgeLen, alpha);
    dxi = -2 * xkEdge * pow(xkjLen, 2) + 2 * pow(xkEdgeLen, 2) * xkj 
        - 2 * (xkEdge * xkj) * (xkEdge - xkj);
    deta = beta * pow(xkjLen, beta-2) * pow(xkEdgeLen, alpha) * xkj
        + alpha * pow(xkjLen, beta) * pow(xkEdgeLen, alpha-2) * (-xkEdge);
}

__device__ void ijk(double* dev_blocks, int p, int q, int r, unsigned int N, double& xi, double&eta, Vector& dxi, Vector& deta, double alpha, double beta)
{
    int i { p };
    int j { q };
    int k { r };
    Vector xkEdge = vectorFromTensor(dev_blocks, k+1, N) - vectorFromTensor(dev_blocks, k, N);
    double xkEdgeLen = xkEdge.norm();
    Vector xij = vectorFromTensor(dev_blocks, i, N) - vectorFromTensor(dev_blocks, j, N);
    double xijLen = xij.norm();

    xi = pow(xkEdgeLen * xijLen, 2) - pow(xkEdge * xij, 2);
    eta = pow(xijLen, beta) * pow(xkEdgeLen, alpha);
    dxi = -2 * xkEdge * pow(xijLen, 2) + 2 * (xkEdge * xij) * xij;
    deta = alpha * pow(xijLen, beta) * pow(xkEdgeLen,alpha-2) * (-xkEdge);
}

__device__ void km1jkm1(double* dev_blocks, int p, int q, int r, unsigned int N, double& xi, double&eta, Vector& dxi, Vector& deta, double alpha, double beta)
{
    int k { p + 1 };
    int j { q };
    Vector xkEdge = vectorFromTensor(dev_blocks, k, N) - vectorFromTensor(dev_blocks, k-1, N);
    double xkEdgeLen = xkEdge.norm();
    Vector xkmj = vectorFromTensor(dev_blocks, k-1, N) - vectorFromTensor(dev_blocks, j, N);
    double xkmjLen = xkmj.norm();

    xi = pow(xkEdgeLen * xkmjLen, 2) - pow(xkEdge * xkmj, 2);
    eta = pow(xkmjLen, beta) * pow(xkEdgeLen, alpha);
    dxi = 2 * pow(xkmjLen, 2) * xkEdge - 2 * (xkEdge * xkmj) * xkmj;
    deta = alpha * pow(xkmjLen, beta) * pow(xkEdgeLen, alpha-2) * xkEdge;
}

__device__ void kjkm1(double* dev_blocks, int p, int q, int r, unsigned int N, double& xi, double&eta, Vector& dxi, Vector& deta, double alpha, double beta)
{
    int k { p };
    int j { q };
    Vector xkEdge = vectorFromTensor(dev_blocks, k, N) - vectorFromTensor(dev_blocks, k-1, N);
    double xkEdgeLen = xkEdge.norm();
    Vector xkj = vectorFromTensor(dev_blocks, k, N) - vectorFromTensor(dev_blocks, j, N);
    double xkjLen = xkj.norm();

    xi = pow(xkEdgeLen * xkjLen, 2) - pow(xkEdge * xkj, 2);
    eta = pow(xkjLen, beta) * pow(xkEdgeLen, alpha);
    dxi = 2 * pow(xkjLen, 2) * xkEdge + 2 * pow(xkEdgeLen, 2) * xkj
        - 2 * (xkEdge * xkj) * (xkEdge + xkj);
    deta = beta * pow(xkEdgeLen, alpha) * pow(xkjLen, beta-2) * xkj
        + alpha * pow(xkjLen, beta) * pow(xkEdgeLen, alpha-2) * xkEdge;
}

__device__ void ikj(double* dev_blocks, int p, int q, int r, unsigned int N, double& xi, double&eta, Vector& dxi, Vector& deta, double alpha, double beta)
{
    int i { p };
    int j { r };
    int k { q };
    Vector xjEdge = vectorFromTensor(dev_blocks, j+1, N) - vectorFromTensor(dev_blocks, j, N);
    double xjEdgeLen = xjEdge.norm();
    Vector xki = vectorFromTensor(dev_blocks, k, N) - vectorFromTensor(dev_blocks, i, N);
    double xkiLen = xki.norm();

    xi = pow(xjEdgeLen * xkiLen, 2) - pow(xjEdge * xki, 2);
    eta = pow(xkiLen, beta) * pow(xjEdgeLen, alpha);
    dxi = 2 * pow(xjEdgeLen, 2) * xki - 2 * (xjEdge * xki) * xjEdge;
    deta = beta * pow(xjEdgeLen, alpha) * pow(xkiLen, beta-2) * xki;
}
