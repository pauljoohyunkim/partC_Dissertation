#include <iostream>
#include "solver.cuh"

CurveTensor::CurveTensor(unsigned int aN)
{
    N = aN;
    hipMalloc((void**)&dev_blocks, 3 * N * sizeof(double));
    std::cout << "Tensor Constructed" << std::endl;
}

CurveTensor::~CurveTensor()
{
    hipFree(dev_blocks);
    std::cout << "Tensor Destructed" << std::endl;
}

void tensorBlockLoad(CurveTensor& Gammabf, double* blocks, unsigned int N)
{
    hipMemcpy(Gammabf.dev_blocks, blocks, 3 * N * sizeof(double), hipMemcpyHostToDevice);
}
void tensorBlockFlush(CurveTensor& Gammabf, double* blocks, unsigned int N)
{
    hipMemcpy(blocks, Gammabf.dev_blocks, 3 * N * sizeof(double), hipMemcpyDeviceToHost);
}
