#include "hip/hip_runtime.h"
#include <iostream>
#include "solver.cuh"

/* (DO NOT USE DIRECTLY) Abstract Function: Addition of Tensor */
__global__ static void cuTensorAdd(CurveTensor& t1, CurveTensor& t2)
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    int offset = i + (int) t1.N * j;

    if (offset < 3 * t1.N)
    {
        t1.dev_blocks[offset] = t1.dev_blocks[offset] + t2.dev_blocks[offset];
    }
}

CurveTensor::CurveTensor(unsigned int aN)
{
    N = aN;
    hipMalloc((void**)&dev_blocks, 3 * N * sizeof(double));
    std::cout << "Tensor Constructed" << std::endl;
}

CurveTensor::~CurveTensor()
{
    hipFree(dev_blocks);
    std::cout << "Tensor Destructed" << std::endl;
}

void tensorBlockLoad(CurveTensor& Gammabf, double* blocks, unsigned int N)
{
    hipMemcpy(Gammabf.dev_blocks, blocks, 3 * N * sizeof(double), hipMemcpyHostToDevice);
}
void tensorBlockFlush(CurveTensor& Gammabf, double* blocks, unsigned int N)
{
    hipMemcpy(blocks, Gammabf.dev_blocks, 3 * N * sizeof(double), hipMemcpyDeviceToHost);
}

void tensorAdd(CurveTensor& t1, CurveTensor& t2)
{
    dim3 grid(3, t1.N);
    cuTensorAdd<<<grid,1>>>(t1, t2);
}

