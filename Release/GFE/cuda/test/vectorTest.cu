#include "hip/hip_runtime.h"
#include <cstdio>
#include "../src/vector.cuh"

__global__ void kernel()
{
    Vector v1 { 0, 1, 2 };
    Vector v2 { 1, -2, 5 };
    auto vp = v1 + v2;
    auto vn = v1 - v2;

    printf("v1: %f, %f, %f\n", v1.x, v1.y, v1.z);
    printf("v2: %f, %f, %f\n", v2.x, v2.y, v2.z);
    printf("sum: %f, %f, %f\n", vp.x, vp.y, vp.z);
    printf("diff: %f, %f, %f\n", vn.x, vn.y, vn.z);
}

int main()
{
    kernel<<<1,1>>>();
    
    hipDeviceSynchronize();
    return 0;
}
