#include "hip/hip_runtime.h"
#include "../src/solver.hpp"
#include <vector>
#include <iostream>

#define LENGTH 4

__global__ void kernel(double* dev_x)
{
    double sum = sumArray(dev_x, LENGTH);

    dev_x[0] = sum;
}

int main()
{
    double arrOfVal[] = { 1, 2, 3, 4 };
    double* dev;
    double sum;

    /* Alloc */
    hipMalloc((void**)&dev, sizeof(double) * LENGTH);
    std::cout << "hipMalloc-ed" << std::endl;

    hipMemcpy(dev, arrOfVal, sizeof(double) * LENGTH, hipMemcpyHostToDevice);
    kernel<<<1, 1>>>(dev);
    hipMemcpy(&sum, dev, sizeof(double), hipMemcpyDeviceToHost);

    std::cout << sum << std::endl;
    
    hipFree(dev);
}
