#include "hip/hip_runtime.h"
#include <iostream>
#include "geometric-objects.hpp"
#include "solver.hpp"

/* Constructor for cuRepulsiveCurve */ 
cuRepulsiveCurve::cuRepulsiveCurve(unsigned int aJ): cuCurve(aJ)
{
    energyMatrixFlattened.resize(aJ * aJ);
}


cuRepulsiveCurve::cuRepulsiveCurve(std::vector<double> &aX, std::vector<double> &aY, std::vector<double> &aZ): cuCurve(aX, aY, aZ)
{
    double aJ = aX.size();
    energyMatrixFlattened.resize(aJ * aJ);
}

/* Deconstructor */
cuRepulsiveCurve::~cuRepulsiveCurve()
{
    if (dev_x_allocated)
    {
        hipFree(dev_x);
    }
    if (dev_y_allocated)
    {
        hipFree(dev_y);
    }
    if (dev_z_allocated)
    {
        hipFree(dev_z);
    }
    if (dev_energyMatrix_allocated)
    {
        hipFree(dev_energyMatrix);
    }

    std::cout << "cuRepulsiveCurve Deallocated" << std::endl;
}

/* Call this function before doing GPU stuff */
void cuRepulsiveCurve::cudafy()
{
    /* Allocate memory and copy data */
    hipMalloc((void**)&dev_x, J * sizeof(double));
    dev_x_allocated = true;
    hipMalloc((void**)&dev_y, J * sizeof(double));
    dev_y_allocated = true;
    hipMalloc((void**)&dev_z, J * sizeof(double));
    dev_z_allocated = true;
    hipMalloc((void**)&dev_energyMatrix, (J * J) * sizeof(double*));
    dev_energyMatrix_allocated = true;

    hipMemcpy(dev_x, &x[0], J * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, &y[0], J * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_z, &z[0], J * sizeof(double), hipMemcpyHostToDevice);

    std::cout << "cuRepulsiveCurve Allocated" << std::endl;
}

void cuRepulsiveCurve::flushFromDevice()
{
    if (!dev_x_allocated)
    {
        throw std::runtime_error("cuRepulsiveCurve::flushFromDevice: dev_x not allocated");
    }
    if (!dev_y_allocated)
    {
        throw std::runtime_error("cuRepulsiveCurve::flushFromDevice: dev_y not allocated");
    }
    if (!dev_z_allocated)
    {
        throw std::runtime_error("cuRepulsiveCurve::flushFromDevice: dev_z not allocated");
    }
    if (!dev_energyMatrix_allocated)
    {
        throw std::runtime_error("cuRepulsiveCurve::flushFromDevice: dev_energyMatrix not allocated");
    }
    hipMemcpy(&x[0], dev_x, J * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&y[0], dev_y, J * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&z[0], dev_z, J * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&energyMatrixFlattened[0], dev_energyMatrix, J * J * sizeof(double), hipMemcpyDeviceToHost);

}

__global__ void fillEnergyMatrix(double* dev_x, double* dev_y, double* dev_z, double* dev_energyMatrix, unsigned int J)
{
    int i = blockIdx.x;
    int ip1 = (i + 1) % J;
    int j = blockIdx.y;
    int jp1 = (j + 1) % J;
    int flattenPos = i + J * j;

    if (abs(i - j) > 1 && abs(i - j + (int) J) > 1 && abs(i - j - (int) J) > 1)
    {
        /* p, q */
        double pix = dev_x[i];
        double piy = dev_y[i];
        double piz = dev_z[i];
        double qix = dev_x[j];
        double qiy = dev_y[j];
        double qiz = dev_z[j];

        /* pI, qJ */
        double pIx = dev_x[ip1] - pix;
        double pIy = dev_y[ip1] - piy;
        double pIz = dev_z[ip1] - piz;
        double qIx = dev_x[jp1] - qix;
        double qIy = dev_y[jp1] - qiy;
        double qIz = dev_z[jp1] - qiz;

        /* lI, lJ */
        double lI = l2norm3D(pIx, pIy, pIz);
        double lJ = l2norm3D(qIx, qIy, qIz);

        /* TI = pI / lI */
        double TIx = pIx / lI;
        double TIy = pIy / lI;
        double TIz = pIz / lI;

        dev_energyMatrix[flattenPos] = kernelFunction(pix, piy, piz, dev_x[ip1], dev_y[ip1], dev_z[ip1],
                qix, qiy, qiz, dev_x[jp1], dev_y[jp1], dev_z[jp1], TIx, TIy, TIz);
    }
    else
    {
        dev_energyMatrix[flattenPos] = 0;
    }
    
}

__device__ double kernelalphabeta(double px, double py, double pz, double qx, double qy, double qz, double Tx, double Ty, double Tz, double alpha, double beta)
{
    double pmqx = px - qx;
    double pmqy = py - qy;
    double pmqz = pz - qz;
    double numx;
    double numy;
    double numz;

    /* T x (p-q) */
    cross(px, py, pz, qx, qy, qz, numx, numy, numz);
    double numerator = pow(l2norm3D(numx, numy, numz), alpha);
    double denominator = pow(l2norm3D(pmqx, pmqy, pmqz), beta);

    return numerator / denominator;
}


__device__ double kernelFunction(double xix, double xiy, double xiz, double xipx, double xipy, double xipz, double xjx, double xjy, double xjz, double xjpx, double xjpy, double xjpz, double Tix, double Tiy, double Tiz)
{
    double kij { 0 };

    kij += kernelalphabeta(xix, xiy, xiz, xjx, xjy, xjz, Tix, Tiy, Tiz, ALPHA, BETA);
    kij += kernelalphabeta(xix, xiy, xiz, xjpx, xjpy, xjpz, Tix, Tiy, Tiz, ALPHA, BETA);
    kij += kernelalphabeta(xipx, xipy, xipz, xjx, xjy, xjz, Tix, Tiy, Tiz, ALPHA, BETA);
    kij += kernelalphabeta(xipx, xipy, xipz, xjpx, xjpy, xjpz, Tix, Tiy, Tiz, ALPHA, BETA);

    return kij / 4;
}

__device__ void cross(double x1, double y1, double z1, double x2, double y2, double z2, double &x3, double &y3, double &z3)
{
    x3 = y1 * z2 - y2 * z1;
    y3 = z1 * x2 - x1 * z2;
    z3 = x1 * y2 - x2 * y1;
}

__device__ double l2norm3D(double x1, double x2, double x3)
{
    double norm { 0 };
    norm += x1 * x1;
    norm += x2 * x2;
    norm += x3 * x3;

    return sqrt(norm);
}
