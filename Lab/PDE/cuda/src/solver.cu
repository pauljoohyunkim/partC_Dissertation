#include "hip/hip_runtime.h"
#include "solver.hpp"
//#include "geometric-objects.hpp"

__device__ double kernelalphabeta(double px, double py, double pz, double qx, double qy, double qz, double Tx, double Ty, double Tz, double alpha, double beta)
{
    double pmqx = px - qx;
    double pmqy = py - qy;
    double pmqz = pz - qz;
    double numx;
    double numy;
    double numz;

    /* T x (p-q) */
    cross(px, py, pz, qx, qy, qz, numx, numy, numz);
    double numerator = pow(l2norm3D(numx, numy, numz), alpha);
    double denominator = pow(l2norm3D(pmqx, pmqy, pmqz), beta);

    return numerator / denominator;
}


__device__ void cross(double x1, double y1, double z1, double x2, double y2, double z2, double &x3, double &y3, double &z3)
{
    x3 = y1 * z2 - y2 * z1;
    y3 = z1 * x2 - x1 * z2;
    z3 = x1 * y2 - x2 * y1;
}

__device__ double l2norm3D(double x1, double x2, double x3)
{
    double norm { 0 };
    norm += x1 * x1;
    norm += x2 * x2;
    norm += x3 * x3;

    return sqrt(norm);
}
